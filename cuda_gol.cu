#include "hip/hip_runtime.h"
#include <ostream>
#include <iostream>
#include <stdio.h>
#include <cstdlib>
#include <stdexcept>

#include "cuda_gol.h"

// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>

__global__ void cuda_gol_update(int *A, int *B, int N)
{
    int k = blockIdx.x;
    int i = threadIdx.x;

    // int r = (k / N);
    // int c = (k % N);

    __shared__ int nbh_c;
    if (threadIdx.x == 0)
        nbh_c = 0;
    __syncthreads();

    int nbh_idx_i, nbh_idx_j;

    nbh_idx_i = ((i % 3) - 1) + (k / N);
    nbh_idx_j = ((i + 3) / 3 - 2) + (k % N);

    if (!((nbh_idx_i < 0 || nbh_idx_j < 0) || (nbh_idx_i >= N || nbh_idx_j >= N) || (nbh_idx_i == k && nbh_idx_j == k)))
    {
        if (A[nbh_idx_i * N + nbh_idx_j] == 1)
        {
            atomicAdd(&nbh_c, 1);
        }
    }
    __syncthreads();

    // for (int i = r - 1; i <= r + 1; i++)
    // {
    //     for (int j = c - 1; j <= c + 1; j++)
    //     {
    //         if ((i == r && j == c) || (i < 0 || j < 0) || (i >= N || j >= N))
    //         {
    //             continue;
    //         }
    //         if (A[i * N + j] == 1)
    //         {
    //             if (threadIdx.x == 0)
    //             nbh_c++;
    //         }
    //     }
    // }

    if (A[k] == 1 && (nbh_c == 2 || nbh_c == 3))
    {
        B[k] = 1;
    }
    else if (A[k] == 0 && nbh_c == 3)
    {
        B[k] = 1;
    }
    else
    {
        B[k] = 0;
    }
}

__global__ void cuda_update_objs(int *A, GameObject *physicsObjs, int N)
{
    int k = blockIdx.x;

    if (A[k] == 0)
    {
        physicsObjs[k].color = {0, 0, 0};
    }
    else
    {
        physicsObjs[k].color = {1, 1, 1};
    }
}

void fct_cuda_stat_init(int *A, int **c_A, int *B, int **c_B, int N)
{
    hipMalloc(c_A, N * N * sizeof(int));
    hipMalloc(c_B, N * N * sizeof(int));

    hipMemcpy(c_A[0], A, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c_B[0], B, N * N * sizeof(int), hipMemcpyHostToDevice);
}

void fct_cuda_stat_updt(int *A, int *c_A, int *B, int *c_B, int N)
{
    cuda_gol_update<<<N * N, 9>>>(c_A, c_B, N);
    hipMemcpy(c_A, c_B, N * N * sizeof(int), hipMemcpyDeviceToDevice);

    // hipMemcpy(A, c_A, N * N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(B, c_B, N * N * sizeof(int), hipMemcpyDeviceToHost);

    memcpy(A, B, N * N * sizeof(int));
}

void fct_cuda_stat_free(int *c_A, int *c_B)
{
    hipFree(c_A);
    hipFree(c_B);
}

void fct_cuda_objs_init(std::vector<GameObject> *p_physicsObjs, GameObject **c_physicsObjs, int N)
{
    hipMalloc(c_physicsObjs, N * N * sizeof(GameObject));
    hipMemcpy(c_physicsObjs[0], p_physicsObjs->data(), N * N * sizeof(GameObject), hipMemcpyHostToDevice);
}

void fct_cuda_objs_updt(int *A, int *c_A, std::vector<GameObject> *p_physicsObjs, GameObject *c_physicsObjs, int N)
{
    cuda_update_objs<<<N * N, 1>>>(c_A, c_physicsObjs, N);

    hipMemcpy(p_physicsObjs->data(), c_physicsObjs, N * N * sizeof(GameObject), hipMemcpyDeviceToHost);
}

void fct_cuda_objs_free(GameObject *c_physicsObjs)
{
    hipFree(c_physicsObjs);
}